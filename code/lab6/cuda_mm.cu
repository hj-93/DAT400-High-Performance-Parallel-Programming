#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#define MATRIX_SIZE 1000
#define GRID_X 200
#define GRID_Y 200
#define GRID_Z 1
#define BLOCK_X 5
#define BLOCK_Y 5
#define BLOCK_Z 1
void printDeviceProp(const hipDeviceProp_t &prop)
{
printf("Device Name : %s.\n", prop.name);
printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
printf("regsPerBlock : %d.\n", prop.regsPerBlock);
printf("warpSize : %d.\n", prop.warpSize);
printf("memPitch : %d.\n", prop.memPitch);
printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
printf("totalConstMem : %d.\n", prop.totalConstMem);
printf("major.minor : %d.%d.\n", prop.major, prop.minor);
printf("clockRate : %d.\n", prop.clockRate);
printf("textureAlignment : %d.\n", prop.textureAlignment);
printf("deviceOverlap : %d.\n", prop.deviceOverlap);
printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

//CUDA Initialization
bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);

    if (count == 0) 
    {
        fprintf(stderr, "There is no device.\n");

        return false;
    }

    int i;

    for (i = 0; i < count; i++) 
    {

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printDeviceProp(prop);

        if (hipGetDeviceProperties(&prop, i) == hipSuccess) 
        {
            if (prop.major >= 1) 
            {
            break;
            }
        }
    }

    if (i == count) 
    {
    fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
    return false;
    }

    hipSetDevice(i);

    return true;

}

// Generate Random Matrix Elements
void matgen(float* a, int n)
{
    int i, j;

    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {

            a[i * n + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);

        }
    }
}

/* Task: Implement Your Kernel Function Here */
__global__ static void matMultCUDA(const float* a, const float* b, float* c, int n)
{
    int g_threadId_x = blockIdx.x * blockDim.x + threadIdx.x;
    int g_threadId_y = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;
    __shared__ float row_vec_shared[BLOCK_X * BLOCK_Y];
    __shared__ float col_vec_shared[BLOCK_X * BLOCK_Y];

    for (int i = 0; i < n/blockDim.x; i++) {
         row_vec_shared[threadIdx.x * blockDim.y + threadIdx.y] = a[g_threadId_x * n + i * blockDim.y + threadIdx.y];
         col_vec_shared[threadIdx.x * blockDim.y + threadIdx.y] = b[(i * blockDim.x + threadIdx.x) * n + g_threadId_y];
         __syncthreads();
         for (int j = 0; j < blockDim.x; j++) {
             c[g_threadId_x * n + g_threadId_y] += row_vec_shared[threadIdx.x * blockDim.y + j] * col_vec_shared[j* blockDim.y + threadIdx.y];
         }
        __syncthreads();
    }
}

int main()
{
    if (!InitCUDA()) return 0; 

    float *a, *b, *c, *d;

    int n = MATRIX_SIZE;

    a = (float*)malloc(sizeof(float)* n * n); 
    b = (float*)malloc(sizeof(float)* n * n); 
    c = (float*)malloc(sizeof(float)* n * n);
    d = (float*)malloc(sizeof(float)* n * n);
    srand(0);

    matgen(a, n);
    matgen(b, n);

    float *cuda_a, *cuda_b, *cuda_c;
    size_t size = sizeof(float) * n * n;
    /* Task: Memory Allocation */
    hipMalloc((void **) &cuda_a, size);
    hipMalloc((void **) &cuda_b, size);
    hipMalloc((void **) &cuda_c, size);
    hipMemset(&cuda_c, 0, size);
    /* Task: CUDA Memory Copy from Host to Device */
    hipMemcpy(cuda_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, size, hipMemcpyHostToDevice);

    /* Task: Number of Blocks and Threads && Dimention*/
    dim3 dimGrid(GRID_X, GRID_Y, GRID_Z);
    dim3 dimBlock(BLOCK_X, BLOCK_Y, BLOCK_Z);

    // Kernel Execution
    matMultCUDA << < dimGrid, dimBlock >> >(cuda_a , cuda_b , cuda_c , n);

    /* Task: CUDA Memory Copy from Device to Host */
    hipMemcpy(c, cuda_c, size, hipMemcpyDeviceToHost);
    
    //Free
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);

    // CPU Implementation of MatMul
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        { 
            double t = 0;

            for (int k = 0; k < n; k++)
            { 

                t += a[i * n + k] * b[k * n + j]; 

            } 

            d[i * n + j] = t; 

        } 
    }

    // Check the accuracy of GPU results with CPU results
    float max_err = 0;
    float average_err = 0; 

    for (int i = 0; i < n; i++) 
    {
        for (int j = 0; j < n; j++) 
        {
            if (d[i * n + j] != 0)
            { 
                float err = fabs((c[i * n + j] - d[i * n + j]) / d[i * n + j]);
                if (max_err < err) max_err = err; 
                average_err += err; 
            } 
        } 
    }

    printf("Max error: %g Average error: %g\n",max_err, average_err / (n * n));
return 0;
}
